#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <chrono>

// 1D convolution kernel with input tiling
__global__ void convolution1D_InputTiling(float *input, float *mask, float *output, 
                                        int inputLength, int maskLength) {
    extern __shared__ float sharedMem[];
    
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int maskRadius = maskLength / 2;
    int tile_size = blockDim.x;
    
    // Load input elements into shared memory with padding for mask radius
    int halo_left_idx = (blockIdx.x * blockDim.x) - maskRadius;
    int halo_right_idx = (blockIdx.x + 1) * blockDim.x + maskRadius - 1;
    int shared_mem_size = tile_size + 2 * maskRadius;
    
    // Load the left halo elements
    if (threadIdx.x < maskRadius) {
        int input_idx = halo_left_idx + threadIdx.x;
        if (input_idx >= 0 && input_idx < inputLength) {
            sharedMem[threadIdx.x] = input[input_idx];
        } else {
            sharedMem[threadIdx.x] = 0.0f;  // Zero padding
        }
    }
    
    // Load the central elements
    int input_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (input_idx < inputLength) {
        sharedMem[threadIdx.x + maskRadius] = input[input_idx];
    } else {
        sharedMem[threadIdx.x + maskRadius] = 0.0f;  // Zero padding
    }
    
    // Load the right halo elements
    if (threadIdx.x < maskRadius) {
        int input_idx = (blockIdx.x + 1) * blockDim.x + threadIdx.x;
        if (input_idx < inputLength) {
            sharedMem[threadIdx.x + tile_size + maskRadius] = input[input_idx];
        } else {
            sharedMem[threadIdx.x + tile_size + maskRadius] = 0.0f;  // Zero padding
        }
    }
    
    // Wait for all threads to finish loading data into shared memory
    __syncthreads();
    
    // Compute convolution using shared memory
    if (idx < inputLength) {
        float result = 0.0f;
        for (int j = 0; j < maskLength; j++) {
            result += sharedMem[threadIdx.x + j] * mask[j];
        }
        output[idx] = result;
    }
}

// Function to read input vector from file
void readInputFile(const char *filename, float **data, int *length) {
    FILE *file = fopen(filename, "r");
    if (!file) {
        fprintf(stderr, "Error opening input file %s\n", filename);
        exit(EXIT_FAILURE);
    }

    fscanf(file, "%d", length);
    *data = (float *)malloc(*length * sizeof(float));
    
    for (int i = 0; i < *length; i++) {
        fscanf(file, "%f", &(*data)[i]);
    }
    
    fclose(file);
}

// Function to read mask from file
void readMaskFile(const char *filename, float **data, int *length) {
    FILE *file = fopen(filename, "r");
    if (!file) {
        fprintf(stderr, "Error opening mask file %s\n", filename);
        exit(EXIT_FAILURE);
    }

    fscanf(file, "%d", length);
    *data = (float *)malloc(*length * sizeof(float));
    
    for (int i = 0; i < *length; i++) {
        fscanf(file, "%f", &(*data)[i]);
    }
    
    fclose(file);
}

// Function to write output data to file
void writeOutputFile(const char *filename, float *data, int length) {
    FILE *file = fopen(filename, "w");
    if (!file) {
        fprintf(stderr, "Error opening output file %s\n", filename);
        exit(EXIT_FAILURE);
    }

    for (int i = 0; i < length; i++) {
        fprintf(file, "%.3f", data[i]);
        if (i < length - 1) {
            fprintf(file, " ");
        }
    }
    fprintf(file, "\n");
    
    fclose(file);
}

int main(int argc, char *argv[]) {
    if (argc != 4) {
        printf("Usage: %s <inputfile> <maskfile> <outputfile>\n", argv[0]);
        return -1;
    }

    const char *inputFile = argv[1];
    const char *maskFile = argv[2];
    const char *outputFile = argv[3];

    // Load input vector from file
    float *h_input;
    int inputLength;
    readInputFile(inputFile, &h_input, &inputLength);

    // Load mask from file
    float *h_mask;
    int maskLength;
    readMaskFile(maskFile, &h_mask, &maskLength);

    // Allocate device memory
    float *d_input, *d_mask, *d_output;
    hipMalloc(&d_input, inputLength * sizeof(float));
    hipMalloc(&d_mask, maskLength * sizeof(float));
    hipMalloc(&d_output, inputLength * sizeof(float));

    // Copy input data and mask to device
    hipMemcpy(d_input, h_input, inputLength * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_mask, h_mask, maskLength * sizeof(float), hipMemcpyHostToDevice);

    // Define block and grid sizes
    int blockSize = 256;
    int gridSize = (inputLength + blockSize - 1) / blockSize;
    
    // Calculate shared memory size for input tiling
    int sharedMemSize = (blockSize + 2 * (maskLength/2)) * sizeof(float);

    // Measure execution time
    auto start = std::chrono::high_resolution_clock::now();

    // Launch convolution kernel with input tiling
    convolution1D_InputTiling<<<gridSize, blockSize, sharedMemSize>>>(d_input, d_mask, d_output, 
                                                                     inputLength, maskLength);
    
    // Wait for GPU to finish
    hipDeviceSynchronize();
    
    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();

    // Copy result back to host
    float *h_output = (float *)malloc(inputLength * sizeof(float));
    hipMemcpy(h_output, d_output, inputLength * sizeof(float), hipMemcpyDeviceToHost);

    // Save result to output file
    writeOutputFile(outputFile, h_output, inputLength);

    printf("Kernel execution time: %.6f ms\n", duration / 1000.0);

    // Free memory
    hipFree(d_input);
    hipFree(d_mask);
    hipFree(d_output);
    free(h_input);
    free(h_mask);
    free(h_output);

    return 0;
}
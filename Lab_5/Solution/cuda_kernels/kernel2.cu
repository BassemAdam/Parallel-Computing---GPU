#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <chrono>
#include <string>
#include <vector>
#include <cstring>

// Define constant memory for mask
__constant__ float c_mask[1024]; // Assuming mask won't exceed 1024 elements

// Structure to store RGB image data
typedef struct {
    unsigned char* data;
    int width;
    int height;
    int channels;
} RGBImage;

// 3D convolution kernel with input tiling
__global__ void convolution3D_InputTiling(
    unsigned char* input,
    unsigned char* output,
    int width, 
    int height, 
    int channels,
    int maskSize,
    int stride
) {
    extern __shared__ unsigned char sharedMem[];
    
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;
    
    int maskRadius = maskSize / 2;
    
    // Calculate the dimensions of the output tile
    int TILE_WIDTH = blockDim.x - maskSize + 1;
    int TILE_HEIGHT = blockDim.y - maskSize + 1;
    
    // Calculate global coordinates of the output pixel
    int outX = bx * TILE_WIDTH + tx - maskRadius;
    int outY = by * TILE_HEIGHT + ty - maskRadius;
    
    // Process each channel separately
    for (int c = 0; c < channels; c++) {
        // Define shared memory for this channel
        unsigned char* sharedData = &sharedMem[c * blockDim.x * blockDim.y];
        
        // Load input element into shared memory with boundary check
        if (outX >= 0 && outX < width && outY >= 0 && outY < height) {
            sharedData[ty * blockDim.x + tx] = input[(outY * width + outX) * channels + c];
        } else {
            // Zero padding for boundary conditions
            sharedData[ty * blockDim.x + tx] = 0;
        }
        
        // Wait for all threads to finish loading their elements
        __syncthreads();
        
        // Only threads that compute output elements will run this
        if (tx >= maskRadius && tx < blockDim.x - maskRadius && 
            ty >= maskRadius && ty < blockDim.y - maskRadius) {
            
            // Calculate output pixel global position
            int outputX = bx * TILE_WIDTH + tx - maskRadius;
            int outputY = by * TILE_HEIGHT + ty - maskRadius;
            
            // Only compute if the output pixel is within image bounds
            if (outputX < width && outputY < height) {
                float sum = 0.0f;
                
                // Apply convolution mask
                for (int ky = 0; ky < maskSize; ky++) {
                    for (int kx = 0; kx < maskSize; kx++) {
                        // Get position in shared memory
                        int sharedX = tx + kx - maskRadius;
                        int sharedY = ty + ky - maskRadius;
                        
                        // Get input pixel from shared memory
                        unsigned char pixel = sharedData[sharedY * blockDim.x + sharedX];
                        
                        // Get the corresponding mask value
                        float maskValue = c_mask[ky * maskSize + kx];
                        
                        // Accumulate weighted sum
                        sum += static_cast<float>(pixel) * maskValue;
                    }
                }
                
                // Clamp results to valid range [0, 255]
                sum = fminf(fmaxf(sum, 0.0f), 255.0f);
                
                // Write result to the output image
                output[(outputY * width + outputX) * channels + c] = static_cast<unsigned char>(sum);
            }
        }
        
        // Wait for all threads to finish using shared memory before next channel
        __syncthreads();
    }
}

// Function to read a 2D mask from file
void readMaskFile(const char* filename, float** mask, int* size) {
    FILE* file = fopen(filename, "r");
    if (!file) {
        fprintf(stderr, "Error opening mask file %s\n", filename);
        exit(EXIT_FAILURE);
    }
    
    // Read mask size (assuming square mask)
    fscanf(file, "%d", size);
    
    // Allocate memory for the mask
    *mask = (float*)malloc((*size) * (*size) * sizeof(float));
    
    // Read mask values
    for (int i = 0; i < (*size) * (*size); i++) {
        fscanf(file, "%f", &((*mask)[i]));
    }
    
    fclose(file);
}

// Load image data using stb_image.h (include this header at the top)
RGBImage loadImage(const char* filename) {
    RGBImage image;
    
    // Placeholder for loading image - you'll need to use a library like stb_image.h
    // For example:
    // image.data = stbi_load(filename, &image.width, &image.height, &image.channels, 3);
    
    printf("Loading image: %s\n", filename);
    
    // This is just a placeholder - replace with actual image loading code
    image.width = 512;   // Example width
    image.height = 512;  // Example height
    image.channels = 3;  // RGB image
    image.data = (unsigned char*)malloc(image.width * image.height * image.channels);
    
    if (!image.data) {
        fprintf(stderr, "Failed to allocate memory for image\n");
        exit(EXIT_FAILURE);
    }
    
    return image;
}

// Save image data using stb_image_write.h (include this header at the top)
void saveImage(const char* filename, RGBImage image) {
    // Placeholder for saving image - you'll need to use a library like stb_image_write.h
    // For example:
    // stbi_write_png(filename, image.width, image.height, image.channels, image.data, image.width * image.channels);
    
    printf("Saving image: %s (%dx%d, %d channels)\n", filename, image.width, image.height, image.channels);
    
    // This is just a placeholder - replace with actual image saving code
}

// Process a batch of images
void processBatch(const std::vector<std::string>& inputFiles, 
                 const std::string& outputFolder,
                 float* h_mask, int maskSize, int stride) {
    
    // Skip if batch is empty
    if (inputFiles.empty()) {
        return;
    }
    
    int batchSize = inputFiles.size();
    printf("Processing batch of %d images\n", batchSize);
    
    // First, load all images in the batch
    std::vector<RGBImage> images;
    std::vector<unsigned char*> d_inputs;
    std::vector<unsigned char*> d_outputs;
    
    for (const auto& inputFile : inputFiles) {
        RGBImage img = loadImage(inputFile.c_str());
        images.push_back(img);
        
        // Allocate device memory for this image
        unsigned char* d_input;
        unsigned char* d_output;
        hipMalloc(&d_input, img.width * img.height * img.channels * sizeof(unsigned char));
        hipMalloc(&d_output, img.width * img.height * img.channels * sizeof(unsigned char));
        
        // Copy image data to device
        hipMemcpy(d_input, img.data, img.width * img.height * img.channels * sizeof(unsigned char), hipMemcpyHostToDevice);
        
        d_inputs.push_back(d_input);
        d_outputs.push_back(d_output);
    }
    
    // Copy mask to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(c_mask), h_mask, maskSize * maskSize * sizeof(float));
    
    // Process each image in the batch
    for (int i = 0; i < batchSize; i++) {
        // Define kernel launch parameters for input tiling
        // Block size includes additional elements for the halo/border region
        int blockWidth = 16 + maskSize - 1;  // Input tile width
        int blockHeight = 16 + maskSize - 1; // Input tile height
        
        dim3 blockSize(blockWidth, blockHeight);
        
        // Output tile dimensions
        int tileWidth = blockWidth - maskSize + 1;  // 16 in this case
        int tileHeight = blockHeight - maskSize + 1; // 16 in this case
        
        // Calculate grid size based on output dimensions
        dim3 gridSize((images[i].width + tileWidth - 1) / tileWidth,
                     (images[i].height + tileHeight - 1) / tileHeight);
        
        // Calculate shared memory size (one tile per channel)
        int sharedMemSize = images[i].channels * blockWidth * blockHeight * sizeof(unsigned char);
        
        // Launch convolution kernel with input tiling
        convolution3D_InputTiling<<<gridSize, blockSize, sharedMemSize>>>(
            d_inputs[i], d_outputs[i],
            images[i].width, images[i].height, images[i].channels,
            maskSize, stride
        );
        
        // Check for kernel launch errors
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(err));
        }
    }
    
    // Wait for all kernels to complete
    hipDeviceSynchronize();
    
    // Copy results back to host and save output images
    for (int i = 0; i < batchSize; i++) {
        // Get output filename
        std::string inputFile = inputFiles[i];
        size_t lastSlash = inputFile.find_last_of("/\\");
        std::string baseName = inputFile.substr(lastSlash + 1);
        std::string outputFile = outputFolder + "/" + baseName;
        
        // Copy processed image back to host
        hipMemcpy(images[i].data, d_outputs[i], images[i].width * images[i].height * images[i].channels * sizeof(unsigned char), hipMemcpyDeviceToHost);
        
        // Save processed image
        saveImage(outputFile.c_str(), images[i]);
        
        // Free device memory
        hipFree(d_inputs[i]);
        hipFree(d_outputs[i]);
        
        // Free host memory
        free(images[i].data);
    }
}

// List all image files in a directory
std::vector<std::string> listImageFiles(const std::string& folderPath) {
    std::vector<std::string> files;
    
    // This is a placeholder - you should use platform-specific code to list files
    // For example, using dirent.h on Linux/macOS or FindFirstFile/FindNextFile on Windows
    
    printf("Listing image files in folder: %s\n", folderPath.c_str());
    
    return files;  // Return list of image files (empty in this placeholder)
}

int main(int argc, char* argv[]) {
    // Check command line arguments
    if (argc < 5 || argc > 6) {
        printf("Usage: %s <input_folder_path> <output_folder_path> <batch_size> <mask_file> [stride]\n", argv[0]);
        return -1;
    }
    
    // Parse command line arguments
    const char* inputFolder = argv[1];
    const char* outputFolder = argv[2];
    int batchSize = atoi(argv[3]);
    const char* maskFile = argv[4];
    int stride = (argc == 6) ? atoi(argv[5]) : 1;  // Default stride is 1
    
    // Print configuration
    printf("Input folder: %s\n", inputFolder);
    printf("Output folder: %s\n", outputFolder);
    printf("Batch size: %d\n", batchSize);
    printf("Mask file: %s\n", maskFile);
    printf("Stride: %d\n", stride);
    
    // Read mask file
    float* h_mask;
    int maskSize;
    readMaskFile(maskFile, &h_mask, &maskSize);
    printf("Mask size: %dx%d\n", maskSize, maskSize);
    
    // List all image files in the input folder
    std::vector<std::string> imageFiles = listImageFiles(inputFolder);
    
    // Process images in batches
    for (int i = 0; i < imageFiles.size(); i += batchSize) {
        // Create a batch of images (up to batchSize)
        std::vector<std::string> batch;
        for (int j = 0; j < batchSize && (i + j) < imageFiles.size(); j++) {
            batch.push_back(imageFiles[i + j]);
        }
        
        // Process this batch
        processBatch(batch, outputFolder, h_mask, maskSize, stride);
    }
    
    // Free host memory for mask
    free(h_mask);
    
    printf("Processing complete.\n");
    return 0;
}